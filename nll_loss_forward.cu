#include "hip/hip_runtime.h"
constexpr int NLL_LOSS_THREADS = 32;

#define AT_DISPATCH_NLL_LOSS_INDEX_TYPES(TYPE, NAME, ...)                   \
  [&] {                                                                     \
    at::ScalarType _it = TYPE;                                              \
    RECORD_KERNEL_FUNCTION_DTYPE(NAME, _it)                                 \
    switch (_it) {                                                          \
      AT_PRIVATE_CASE_TYPE_USING_HINT(NAME, at::ScalarType::Byte, uint8_t, index_t, __VA_ARGS__) \
      AT_PRIVATE_CASE_TYPE_USING_HINT(NAME, at::ScalarType::Long, int64_t, index_t, __VA_ARGS__)\
      default:                                                              \
        AT_ERROR(#NAME, " not implemented for '", toString(_it), "'");      \
    }                                                                       \
  }()

//template <typename scalar_t, typename accscalar_t, typename index_t>
__global__ void nll_loss_forward_reduce_cuda_kernel_2d(
    float* output,
    float* total_weight,
    float* input,
    long* target,
    float* weights,
    bool size_average,
    int nframe,
    int ndim,
    int n_classes,
    int64_t ignore_index) {
  // NOLINTNEXTLINE(cppcoreguidelines-init-variables)
  __shared__ float sh_inputs[NLL_LOSS_THREADS],
      acc_weight[NLL_LOSS_THREADS];

  sh_inputs[threadIdx.x] = static_cast<float>(0);
  acc_weight[threadIdx.x] = static_cast<float>(0);
  for (int i = threadIdx.x; i < nframe; i += NLL_LOSS_THREADS) {
    int t = target[i];
    if (t != static_cast<int>(ignore_index)) {
      CUDA_KERNEL_ASSERT(t >= 0 && t < n_classes);
      scalar_t cur_weight =
          weights != nullptr ? weights[t] : static_cast<float>(1);
      sh_inputs[threadIdx.x] -= input[i * ndim + t] * cur_weight;
      acc_weight[threadIdx.x] += cur_weight;
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    accscalar_t output_acc = 0;
    accscalar_t total_weight_acc = 0;
    for (int i = 0; i < NLL_LOSS_THREADS; ++i) {
      output_acc += sh_inputs[i];
      total_weight_acc += acc_weight[i];
    }
    *total_weight = static_cast<float>(total_weight_acc);
    if (size_average) {
      *output = static_cast<float>(output_acc / total_weight_acc);
    } else {
      *output = static_cast<float>(output_acc);
    }
  }
}
