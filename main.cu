#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <hip/hip_runtime_api.h>

#include <getopt.h>


extern "C" __global__ void dense_2048x768x2304_rtx_2070(float* __restrict__ X, float* __restrict__ W, float* __restrict__ T_matmul_NT);

#define BENCHMARK(i) \
void benchmark_##i(float *const X, float *const W, float *const Y, \
                   const int B, const int I, const int H, \
                   const bool is_nvprof_enabled)

BENCHMARK(0);
BENCHMARK(1);
BENCHMARK(2);


int main(int argc, char *argv[]) {
  int i = -1;  // benchmark to run
  int T = -1;
  int I = -1;
  int H = -1;
  bool is_nvprof_enabled = false;
  int opt;

  while ((opt = getopt(argc, argv, "i:T:I:H:p")) != -1) {
    switch (opt) {
    case 'i':
      std::cout << "Benchmark #" << optarg << std::endl;
      i = std::atoi(optarg);
      continue;
    case 'T':
      T = std::atoi(optarg);
      continue;
    case 'I':
      I = std::atoi(optarg);
      continue;
    case 'H':
      H = std::atoi(optarg);
      continue;
    case 'p':
      std::cout << "Enabling NVProf" << std::endl;
      is_nvprof_enabled = true;
      continue;
    default:
      exit(EXIT_FAILURE);
    }
  }

  assert(i != -1 && "The benchmark index MUST be provided");

  if (i == 0 || i == 2) {
    T = 128;
    I = 768;
    H = 2304;
  } else if (i == 1) {
    I = 768;
    H = 2304;
  }

  const int B = 16 * T;
  const int MaxB = 16 * 128;
  assert(argc != 1 && "Size of argument must be equal to 1");
  const int MaxI = 3072;
  const int MaxH = 3072;
  std::cout << "T=" << T << std::endl;
  std::cout << "I=" << I << std::endl;
  std::cout << "H=" << H << std::endl;
  assert(T > 0 && I > 0 && H > 0 && "The parameters must be provided");
  assert(B <= MaxB && I <= MaxI && H <= MaxH &&
         "The parameters must be smaller than the allowed maximum value");

  float *X, *W, *Y;

  hipMalloc(&X, sizeof(float) * MaxB * MaxI);
  hipMalloc(&W, sizeof(float) * MaxH * MaxI);
  hipMalloc(&Y, sizeof(float) * MaxB * MaxH);

#define CALL_BENCHMARK(num)                                                     \
if (i == num) benchmark_##num(X, W, Y, B, I, H, is_nvprof_enabled)

  CALL_BENCHMARK(0);  // ./main.exe -i 0
  CALL_BENCHMARK(1);  // ./main.exe -i 1 -T 128
  CALL_BENCHMARK(2);  // ./main.exe -i 2
  return 0;
}


inline int floordiv(int a, int b) {
  return a / b;
}


#define TIMER_BEGIN(BlockName)                                                  \
float elapsedTime##BlockName = 0.;                                              \
{                                                                               \
  auto tic = std::chrono::system_clock::now();

#define TIMER_END(BlockName)                                                    \
  auto toc = std::chrono::system_clock::now();                                  \
  elapsedTime##BlockName =                                                      \
      std::chrono::duration_cast<std::chrono::microseconds>(toc - tic).count()  \
      * 1.0;                                                                    \
  std::cout << "ElapsedTime (us)=" << elapsedTime##BlockName << std::endl;      \
}


class CUDAFunctionWrapper {
private:
  const std::function<void(void)> f;
  const float FLOPs;
  const bool is_nvprof_enabled;
public:
  CUDAFunctionWrapper(const std::function<void(void)> f, const float FLOPs,
                      const bool is_nvprof_enabled)
      : f(f), FLOPs(FLOPs), is_nvprof_enabled(is_nvprof_enabled) {}
  void operator()() const {
    if (is_nvprof_enabled) {
      f();
      return;
    }
    for (int i = 0; i < 1000; ++i) {  // warmup run
      f();
    }
    checkCudaErrors(hipDeviceSynchronize());
    TIMER_BEGIN();
    for (int i = 0; i < 1000; ++i) {
      f();
    }
    checkCudaErrors(hipDeviceSynchronize());
    TIMER_END();
    std::cout << "TFLOPS=" << FLOPs / elapsedTime / 1e3 << std::endl;
  }
};


BENCHMARK(0) {
  size_t grid_size = B * H / 128;
  std::cout << "pading<<<" << grid_size << ", 128>>>"
            << std::endl;
  auto f = [&]() {
             hipMemcpy(X, Y, sizeof(float) * B * H, hipMemcpyDeviceToDevice);
           };
  CUDAFunctionWrapper wrapper(f, B * H, is_nvprof_enabled);
  wrapper();
}


BENCHMARK(1) {
  size_t grid_size = B * H / 128 / 64;
  std::cout << "dense_2048x768x2304_rtx_2070<<<" << grid_size << ", 64>>>"
            << std::endl;
  auto f = [&]() {
             dense_2048x768x2304_rtx_2070<<<grid_size, 64>>>(X, W, Y);
           };
  CUDAFunctionWrapper wrapper(f, 2. * B * I * H, is_nvprof_enabled);
  wrapper();
}


BENCHMARK(2) {
  {
    auto f = [&]() {
               dense_2048x768x2304_rtx_2070<<<576, 64>>>(X, W, Y);
             };
    CUDAFunctionWrapper wrapper(f, 2. * 16 * 128 * I * H, is_nvprof_enabled);
    wrapper();
  }
  {
    auto f = [&]() {
               dense_2048x768x2304_rtx_2070<<<540, 64>>>(X, W, Y);
             };
    CUDAFunctionWrapper wrapper(f, 2. * 16 * 120 * I * H, is_nvprof_enabled);
    wrapper();
  }
  {
    auto f = [&]() {
               dense_2048x768x2304_rtx_2070<<<576, 64>>>(X, W, Y);
             };
    CUDAFunctionWrapper wrapper(f, 2. * 16 * 128 * I * H, is_nvprof_enabled);
    wrapper();
  }
}
