#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <hip/hip_runtime_api.h>

#include <getopt.h>


//template <typename scalar_t, typename accscalar_t, typename index_t>
extern "C" __global__ void nll_loss_forward_reduce_cuda_kernel_2d(float* output,
    float* total_weight,
    float* input,
    long* target,
    float* weights,
    bool size_average,
    int nframe,
    int ndim,
    int n_classes,
    int64_t ignore_index);

#define BENCHMARK(i) \
void benchmark_##i(float *const X, float *const W, float *const Y, \
                   const int B, const int I, const int H, \
                   const bool is_nvprof_enabled, long *const Z, float *const A, bool const R, int const L, int const J, int const K, int64_t const P)

BENCHMARK(0);
BENCHMARK(1);
BENCHMARK(2);


int main(int argc, char *argv[]) {
  int i = -1;  // benchmark to run
  int T = -1;
  int I = -1;
  int H = -1;
  bool is_nvprof_enabled = false;
  int opt;

  while ((opt = getopt(argc, argv, "i:T:I:H:p")) != -1) {
    switch (opt) {
    case 'i':
      std::cout << "Benchmark #" << optarg << std::endl;
      i = std::atoi(optarg);
      continue;
    case 'T':
      T = std::atoi(optarg);
      continue;
    case 'I':
      I = std::atoi(optarg);
      continue;
    case 'H':
      H = std::atoi(optarg);
      continue;
    case 'p':
      std::cout << "Enabling NVProf" << std::endl;
      is_nvprof_enabled = true;
      continue;
    default:
      exit(EXIT_FAILURE);
    }
  }

  assert(i != -1 && "The benchmark index MUST be provided");

  if (i == 0 || i == 2) {
    T = 128;
    I = 768;
    H = 2304;
  } else if (i == 1) {
    I = 768;
    H = 2304;
  }

  const int B = 16 * T;
  const int MaxB = 16 * 128;
  assert(argc != 1 && "Size of argument must be equal to 1");
  const int MaxI = 3072;
  const int MaxH = 3072;
  std::cout << "T=" << T << std::endl;
  std::cout << "I=" << I << std::endl;
  std::cout << "H=" << H << std::endl;
  assert(T > 0 && I > 0 && H > 0 && "The parameters must be provided");
  assert(B <= MaxB && I <= MaxI && H <= MaxH &&
         "The parameters must be smaller than the allowed maximum value");
    
    float *X, *W, *Y;
    long *Z;
    
    float *A;
    bool R;
    int L, K, J;
    int64_t P;
    
 

  hipMalloc(&X, sizeof(float) * MaxB * MaxI);
  hipMalloc(&W, sizeof(float) * MaxH * MaxI);
  hipMalloc(&Y, sizeof(float) * MaxB * MaxH);
    
  hipMalloc(&Z, sizeof(long) * MaxB * MaxH);
  
  hipMalloc(&A, sizeof(float) * MaxI * MaxH);
    
  
  R = true;
    
  L = MaxB * MaxI;
  K =  MaxB * MaxH;
  J = MaxH * MaxI;
  
    
  P = static_cast<int64_t>(MaxB * MaxI);
  

#define CALL_BENCHMARK(num)                                                     \
if (i == num) benchmark_##num(X, W, Y, B, I, H, is_nvprof_enabled, Z, A,R, L, K, J, P)

  CALL_BENCHMARK(0);  // ./main.exe -i 0
  CALL_BENCHMARK(1);  // ./main.exe -i 1 -T 128
  CALL_BENCHMARK(2);  // ./main.exe -i 2
  return 0;
}


inline int floordiv(int a, int b) {
  return a / b;
}


#define TIMER_BEGIN(BlockName)                                                  \
float elapsedTime##BlockName = 0.;                                              \
{                                                                               \
  auto tic = std::chrono::system_clock::now();

#define TIMER_END(BlockName)                                                    \
  auto toc = std::chrono::system_clock::now();                                  \
  elapsedTime##BlockName =                                                      \
      std::chrono::duration_cast<std::chrono::microseconds>(toc - tic).count()  \
      * 1.0;                                                                    \
  std::cout << "ElapsedTime (us)=" << elapsedTime##BlockName << std::endl;      \
}


class CUDAFunctionWrapper {
private:
  const std::function<void(void)> f;
  const float FLOPs;
  const bool is_nvprof_enabled;
public:
  CUDAFunctionWrapper(const std::function<void(void)> f, const float FLOPs,
                      const bool is_nvprof_enabled)
      : f(f), FLOPs(FLOPs), is_nvprof_enabled(is_nvprof_enabled) {}
  void operator()() const {
    if (is_nvprof_enabled) {
      f();
      return;
    }
    for (int i = 0; i < 1000; ++i) {  // warmup run
      f();
    }
    checkCudaErrors(hipDeviceSynchronize());
    TIMER_BEGIN();
    for (int i = 0; i < 1000; ++i) {
      f();
    }
    checkCudaErrors(hipDeviceSynchronize());
    TIMER_END();
    std::cout << "TFLOPS=" << FLOPs / elapsedTime / 1e3 << std::endl;
  }
};


BENCHMARK(0) {
  size_t grid_size = B * H / 128;
  std::cout << "pading<<<" << grid_size << ", 128>>>"
            << std::endl;
  auto f = [&]() {
             hipMemcpy(X, Y, sizeof(float) * B * H, hipMemcpyDeviceToDevice);
           };
  CUDAFunctionWrapper wrapper(f, B * H, is_nvprof_enabled);
  wrapper();
}


BENCHMARK(1) {
    
  size_t grid_size = B * H / 128 / 64;
  std::cout << "nll_loss_forward<<<" << grid_size << ", 64>>>"
            << std::endl;
  auto f = [&]() {
             nll_loss_forward_reduce_cuda_kernel_2d<<<grid_size, 64>>>(X, W, Y, Z, A, R, L, K, J, P);
           };
  CUDAFunctionWrapper wrapper(f, 2. * B * I * H, is_nvprof_enabled);
  wrapper();
}


BENCHMARK(2) {
  {
    auto f = [&]() {
               nll_loss_forward_reduce_cuda_kernel_2d<<<576, 64>>>(X, W, Y, Z, A, R, L, K, J, P);
             };
    CUDAFunctionWrapper wrapper(f, 2. * 16 * 128 * I * H, is_nvprof_enabled);
    wrapper();
  }
  {
    auto f = [&]() {
               nll_loss_forward_reduce_cuda_kernel_2d<<<540, 64>>>(X, W, Y, Z, A, R, L, K, J, P);
             };
    CUDAFunctionWrapper wrapper(f, 2. * 16 * 120 * I * H, is_nvprof_enabled);
    wrapper();
  }
  {
    auto f = [&]() {
               nll_loss_forward_reduce_cuda_kernel_2d<<<576, 64>>>(X, W, Y, Z, A, R, L, K, J, P);
             };
    CUDAFunctionWrapper wrapper(f, 2. * 16 * 128 * I * H, is_nvprof_enabled);
    wrapper();
  }
}
